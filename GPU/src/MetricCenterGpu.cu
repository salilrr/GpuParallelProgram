
#include <hip/hip_runtime.h>
//****************************
//File:MetricCenterGpu.cu
//author:Salil Rajadhyaksha
//version:1-Nov-2015
//***************************

//number of thread per block
#define NT 1024

//Structue for 2-D points
typedef struct
{
double x;
double y;
}vector_t;

//Structure for result to store radius and index.
typedef struct
{
double radius;
int index;
}ResultTuple;

//per thread variable in shared memory to store the max dist found by each thread
__shared__ ResultTuple points[NT];

/**
 * Calculate the euclidean distance between two points.
 *
 * @param  a :Index of first point.
 * @param  b  :Index of second point.
 * @param   xy :array that holds the points.
 * @return the euclidean distance.
 */
__device__ double calculateEuclideanDistance(int a,int b,vector_t *xy)
{
	vector_t *pointA=&xy[a];
	vector_t *pointB=&xy[b];
	double dx=pointA->x - pointB->x;
	double dy=pointA->y - pointB->y;
	return sqrt(dx*dx+dy*dy);	
}
/*
find maximum of the two points by comparing their radius
@param a: first point;
@param b: secon point;
@return a:The max of a and b stored in a;
*/
__device__ ResultTuple *findMaxDistance( ResultTuple *a ,ResultTuple *b)
{
	if(a->radius < b->radius)	
		a->radius=b->radius;
		
	return a;
}

/*
Return  the minimum of the two points.
@param :a first point;
@param :b second point;

@returns :a:the minimum stored in a;
*/
__device__ ResultTuple *reduce(ResultTuple *a, ResultTuple *b)
{
	if(a->radius>b->radius||a->radius==-1)
	{
		a->radius=b->radius;
		a->index=b->index;
	}
	
	return a;
}

/**

return maximum between two doubles

@param x: pointer to first double value;
@param y: second value
@return max stored in x;
*/
__device__ double *returnMax(double *x,double y)
{
	if(*x<y)
	*x=y;	 
	return x;
 }
/**
Device Kernel to find the smallest distance between a set of points per block.
Called with a one dimensional grid.
Blocks= number of multiprocessors.
thread=1024 per block.
Each block calculates max distance for one point at a time in a for scheduled in leapfrog.
The threads calculate the max distance for that one point in the inner for schdeuled again in leapfrog.

@param :xy-List of points 
@param :N- total number of points.
@param :result- the array to store the semi-final result per block;
*/
extern "C" __global__ void calculateRadius(vector_t *xy,int N,ResultTuple* result)
{
int thr,bID,numberOfBlocks;
double max;
thr=threadIdx.x;
bID=blockIdx.x;
numberOfBlocks=gridDim.x;

for(unsigned long long int i=bID;i<=N;i+=numberOfBlocks)//schedule points to blocks in leapfrog pattern.
	{						  
		max=0.00;	
		for(unsigned long long int j=thr;j<N;j+=NT)//calculate distance of for current point with respect to all points in leapfrog.
			{
				if(j==i)
					continue;	
				returnMax(&max,calculateEuclideanDistance(i,j,xy));	//call to find the max passing the indices of points and current max.		
			}
		points[thr]=(ResultTuple){max,i};//storing the max in the shared memory.
		
__syncthreads();
//calculate the maximum for that point via shared memory parallel reduction.
   for (int k =NT/2;k>0;k>>=1)
      {
      if (thr<k)
         {
			findMaxDistance(&points[thr],&points[thr+k]);
         }
__syncthreads();
      }
	  //single threaded section.
	if(thr==0)
	{
		reduce(&result[bID],&points[thr]);//reduce to store if current point is less than the min so far for this block.
	}

	}
	
}